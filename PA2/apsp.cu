#include "hip/hip_runtime.h"
// PLEASE MODIFY THIS FILE TO IMPLEMENT YOUR SOLUTION

// Brute Force APSP Implementation:

#include "apsp.h"
#define MAX_P_SIZE 32

namespace {
__global__ void diag_floyed(int n, int p, int p_size, int *graph) {
    int i = p * p_size + threadIdx.y;
    int j = p * p_size + threadIdx.x;

    __shared__ int shared_int[MAX_P_SIZE][MAX_P_SIZE + 1];
    if (i < n && j < n) shared_int[threadIdx.y][threadIdx.x] = graph[i * n + j];
    else shared_int[threadIdx.y][threadIdx.x] = INT_MAX;
    __syncthreads();

    for (int k = p * p_size, k_off = 0; k < (p + 1) * p_size; ++k, ++k_off) {
        if (i < n && j < n && k < n) 
            shared_int[threadIdx.y][threadIdx.x] = min(shared_int[threadIdx.y][threadIdx.x], shared_int[k_off][threadIdx.x] + shared_int[threadIdx.y][k_off]);
        __syncthreads();
    }
    if (i < n && j < n) {
        graph[i * n + j] = shared_int[threadIdx.y][threadIdx.x];
    }
    __syncthreads();
}

__global__ void cross_floyed(int n, int p, int p_size, int *graph) {
    if (blockIdx.x == p) return;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    bool is_row = (blockIdx.y == 1);
    int i = is_row ? p * p_size + ty : blockIdx.x * p_size + ty;
    int j = is_row ? blockIdx.x * p_size + tx : p * p_size + tx;

    __shared__ int shared_pivot_row[MAX_P_SIZE][MAX_P_SIZE + 1];
    __shared__ int shared_pivot_col[MAX_P_SIZE][MAX_P_SIZE + 1];

    if ((p * p_size + ty) < n && j < n)
        shared_pivot_col[tx][ty] = graph[(p * p_size + ty) * n + j];
    else
        shared_pivot_col[tx][ty] = INT_MAX;
    if (i < n && (p * p_size + tx) < n)
        shared_pivot_row[ty][tx] = graph[i * n + p * p_size + tx];
    else
        shared_pivot_row[ty][tx] = INT_MAX;

    __syncthreads();

    if (i < n && j < n) {
        int val = graph[i * n + j];

        for (int k = 0; k < p_size; ++k) if (p * p_size + k < n) {
            int temp;
            if (is_row)
                temp = shared_pivot_row[ty][k] + shared_pivot_col[tx][k];
            else
                temp = shared_pivot_row[ty][k] + shared_pivot_col[tx][k];
            val = min(val, temp);
        }

        graph[i * n + j] = val;
    }
}

// __global__ void cross_floyed_col(int n, int p, int p_size, int *graph) {
//     if (blockIdx.x == p) return;
//     int i = blockIdx.x * p_size + threadIdx.y;
//     int j = p * p_size + threadIdx.x;
//     if (i >= n || j >= n) return;

//     __shared__ int shared_col[MAX_P_SIZE];

//     for (int k = p * p_size; k < (p + 1) * p_size; ++k) {
//         if (k >= n) break;

//         if (threadIdx.y == 0) shared_col[threadIdx.x] = graph[k * n + j];
//         __syncthreads();

//         int d_ik = graph[i * n + k];
//         int d_kj = shared_col[threadIdx.x];
//         int &d_ij = graph[i * n + j];
//         if (d_ik + d_kj < d_ij)
//             d_ij = d_ik + d_kj;
//         __syncthreads();
//     }
// }

// __global__ void cross_floyed_row(int n, int p, int p_size, int *graph) {
//     if (blockIdx.x == p) return;
//     int i = p * p_size + threadIdx.y;
//     int j = blockIdx.x * p_size + threadIdx.x;
//     if (i >= n || j >= n) return;

//     __shared__ int shared_row[MAX_P_SIZE];

//     for (int k = p * p_size; k < (p + 1) * p_size; ++k) {
//         if (k >= n) break;

//         if (threadIdx.x == 0) shared_row[threadIdx.y] = graph[i * n + k];
//         __syncthreads();

//         int d_ik = shared_row[threadIdx.y];
//         int d_kj = graph[k * n + j];
//         int &d_ij = graph[i * n + j];
//         if (d_ik + d_kj < d_ij)
//             d_ij = d_ik + d_kj;
//         __syncthreads();
//     }
// }

__global__ void others_floyed(int n, int p, int p_size, int *graph) {
    if (blockIdx.y == p || blockIdx.x == p) return;
    int i = blockIdx.y * p_size + threadIdx.y;
    int j = blockIdx.x * p_size + threadIdx.x;
    if (i >= n || j >= n) return;

    __shared__ int shared_row[MAX_P_SIZE];
    __shared__ int shared_col[MAX_P_SIZE];

    for (int k = p * p_size; k < (p + 1) * p_size; ++k) {
        if (k >= n) break;

        shared_row[threadIdx.y] = graph[i * n + k];
        shared_col[threadIdx.x] = graph[k * n + j];
        __syncthreads();

        int d_ik = shared_row[threadIdx.y];
        int d_kj = shared_col[threadIdx.x];
        int &d_ij = graph[i * n + j];
        if (d_ik + d_kj < d_ij)
            d_ij = d_ik + d_kj;

        __syncthreads();
    }
}

}

void apsp(int n, /* device */ int *graph) {
    int blk_size = (n - 1) / MAX_P_SIZE + 1;
    for (int p = 0; p * MAX_P_SIZE < n; p++) {
        dim3 thr(MAX_P_SIZE, MAX_P_SIZE);
        diag_floyed<<<1, thr>>>(n, p, MAX_P_SIZE, graph);
        hipDeviceSynchronize();
        // cross_floyed_col<<<blk_size, thr>>>(n, p, MAX_P_SIZE, graph);
        // cross_floyed_row<<<blk_size, thr>>>(n, p, MAX_P_SIZE, graph);
        // hipDeviceSynchronize();
        dim3 blk1(blk_size, 2);
        cross_floyed<<<blk1, thr>>>(n, p, MAX_P_SIZE, graph);
        hipDeviceSynchronize();
        dim3 blk(blk_size, blk_size);
        others_floyed<<<blk, thr>>>(n, p, MAX_P_SIZE, graph);
        hipDeviceSynchronize();
    }
}

// namespace {

// __global__ void kernel(int n, int k, int *graph) {
//     auto i = blockIdx.y * blockDim.y + threadIdx.y;
//     auto j = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < n && j < n && i != k && j != k) {
//         graph[i * n + j] = min(graph[i * n + j], graph[i * n + k] + graph[k * n + j]);
//     }
// }

// }

// void apsp(int n, /* device */ int *graph) {
//     for (int k = 0; k < n; k++) {
//         dim3 thr(32, 32);
//         dim3 blk((n - 1) / 32 + 1, (n - 1) / 32 + 1);
//         kernel<<<blk, thr>>>(n, k, graph);
//     }
// }
    
    